#include "hip/hip_runtime.h"
#include "rendering.h"

__device__ float2 csqr(float2 z){
    return float2(z.x*z.x - z.y*z.y, 2*z.x*z.y);
}

__device__ float4 Color(float m){
    m = min(m, 1000.0);
    float s = 2*(1.0 / (1 + exp(-m)) - 0.5);
    return float4(0, 1.0-s, fmod(s+0.5f, 1.0f), 1.0f);
}

struct MandelbrotInfo{
    int N;
    float_2 C;
};

extern "C" __global__
void kernel(int width, int height, ConstantBuffer<MandelbrotInfo> &info, Texture2D<float4> &render_target) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int py = tid / width;
    int px = tid % width;
    if (py >= height)
    return;

    float2 Z = float2((px + 0.5)/width, (py + 0.5)/height) * 2.0 - 1.0;

    for (int i=0; i<info->N; i++)
        Z = csqr(Z) + info->C;

    render_target.Save(int2(px, py), Color(dot(Z, Z)));
}
